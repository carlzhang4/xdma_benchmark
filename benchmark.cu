#include "hip/hip_runtime.h"
#include "benchmark.cuh"

#define ErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void write_bridge_gpu(volatile unsigned int* bridge, size_t length){
	int index = threadIdx.x;
	int iteration_time = length/64;
	__syncthreads();
	if(index<16){
		for(int i=0;i<iteration_time;i++){
			bridge[index + 16*i] = index+64;	
		}
	}
	__syncthreads();
}




void gpu_test(void* bridge){
	void * addr = (void*)bridge;
	size_t length = 512*1024;
	hipError_t cudaerr = hipHostRegister((void*)addr,length,hipHostRegisterIoMemory);
	ErrCheck(cudaerr);
	unsigned int * dev_addr;
	cudaerr = hipHostGetDevicePointer((void **) &(dev_addr),addr, 0);
	ErrCheck(cudaerr);

	write_bridge_gpu <<<1,16>>> (dev_addr, length);
	hipDeviceSynchronize();
	cudaerr = hipPeekAtLastError();
	ErrCheck(cudaerr);

	
}